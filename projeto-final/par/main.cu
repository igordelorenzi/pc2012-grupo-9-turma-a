#include "hip/hip_runtime.h"
/**
* compilar: nvcc main.cu -o main -arch sm_11
* executar: sh exec.sh <num_blocos> <num_threads>
**/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

// Tamanho da palavra
#define PTAM 5
// Define Palavra como array de char de 6 posições
typedef char Palavra[PTAM + 1];

/*
* Estrutura do dicionário
* Guarda palavra e se foi encontrada	
*/
struct palavraStruct
{
	Palavra pal;
	unsigned int encontrado;
};

__global__
void semearCuda(hiprandState *estado, int semente);
__host__ __device__
int comparaCuda(Palavra p1, Palavra p2);
__device__
void geraPalavraCuda(Palavra palGerada, unsigned int tam, hiprandState* estado);
__device__
void verificaPalavraCuda(Palavra palGerada, palavraStruct *dic, unsigned int dicTam, unsigned int* cont);
__global__
void processoCuda(palavraStruct *dic, unsigned int dicTam, hiprandState *estado, unsigned int *cont);
int nrLinhas(char *arquivo);
int comparaQuick(const void *a, const void *b);

int main(int argc,char **argv)
{
	// Variaveis host
	timeval tempo;
	int blocks, threads, porcentagem;
	unsigned int i;
	FILE* in;
	char palavra[PTAM + 1];
	palavraStruct *hDic;
	hiprandState *hEst;
	unsigned int hTam;	
	unsigned int hCont = 0;

	// Variaveis device
	palavraStruct *dDic;
	hiprandState *dEst;
	unsigned int *dCont;
	// Verifica argumentos
	if(argc != 4)
    	{
        	printf("Uso:\n\t%s <arquivo_filtrado> <num_blocos> <num_threads>\n",argv[0]);
		return -1;
    	}
	// Converte parametros de blocos e threads
	blocks = atoi(argv[2]);
	threads = atoi(argv[3]);
	// Le o numero de palavras no dicionario
	hTam = nrLinhas(argv[1]);
	// Aloca vetor de estados no host
	hEst = (hiprandState *)malloc(threads * blocks * sizeof(hiprandState));

	printf("Lendo palavras filtradas...");
	// Aloca dicionario no host
	hDic = (palavraStruct *) malloc(hTam * sizeof(palavraStruct));

	// Povoa o dicionario com as palavras do arquivo filtrado	
	in = fopen(argv[1],"r");
	i = 0;
	while (fscanf(in, "%s", palavra) != EOF) {
		strcpy(hDic[i].pal, palavra);
		hDic[i].encontrado = 0;
		i++;
	}
	fclose(in);
	printf("OK!\n");

	// Ordena o dicionario em memória
	printf("Ordenando as palavras...");
	qsort(hDic, hTam , sizeof(struct palavraStruct), comparaQuick); 
	printf("OK!\n");
	
	// Aloca as variaveis em device
	hipMalloc((void **)&dEst, threads * blocks * sizeof(hiprandState));
	hipMalloc((void **)&dDic, sizeof(palavraStruct)*hTam);
	hipMalloc((void **)&dCont, sizeof(unsigned int));

	// Inicializa variavel contador em device	
	hipMemset(dCont, 0, sizeof(unsigned int));

	// Copia os dados do host para device
	hipMemcpy(dDic, hDic, sizeof(palavraStruct)*hTam, hipMemcpyHostToDevice);
	hipMemcpy(dEst, hEst, sizeof(hiprandState)*threads * blocks, hipMemcpyHostToDevice);	
	semearCuda<<<blocks, threads>>>(dEst, time(NULL));
	
	porcentagem = 10;
	gettimeofday(&tempo, NULL);
	double t1 = tempo.tv_sec + (tempo.tv_usec/1000000.0);

	// Laço principal, para quando encontra todas palavras do dicionario
	while(hCont < hTam)
	{
		// Pula pelo numero total de threads trabalhando
		for(i = 0 ; i < hTam ; i += blocks * threads)
		{
			// Chama processo para cada threads de todos os blocos
			processoCuda<<<blocks, threads>>>(dDic, hTam, dEst, dCont);
	    	}
		// Copia contador de device para host, para verificacao
		hipMemcpy(&hCont, dCont, sizeof(unsigned int), hipMemcpyDeviceToHost);
		// Verifica a porcentagem ja encontrada
		if((hCont*100) / hTam >= porcentagem)
		{
			gettimeofday(&tempo, NULL);
			double t2 = tempo.tv_sec + (tempo.tv_usec/1000000.0);
			printf("%d %% (%.3lf segundos)\n", porcentagem, t2-t1);
			porcentagem += 10;
		}	
	}
	// Copia o dicionario para o host novamente	
	hipMemcpy(hDic, dDic, sizeof(palavraStruct)*hTam, hipMemcpyDeviceToHost);
	
	// Libera memoria em device e no host
	free(hDic);
	free(hEst);
	hipFree(dEst);
	hipFree(dDic);
	hipFree(dCont);

	return 0;
}

/*
* Cria uma semente para cada thread
*/
__global__
void semearCuda(hiprandState *estado, int semente)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	hiprand_init(semente + idx, 0, 0, &estado[idx]);
}

/*
* Verifica na GPU se palavra gerada é igual do dicionário
*/
__host__ __device__
int comparaCuda(Palavra p1, Palavra p2)
{
	unsigned int i;
	for(i = 0 ; i < PTAM ; i++)
	{
		if(p1[i] != p2[i] || p1[i] == '\0' || p2[i] == '\0')
	    		return p1[i] - p2[i];
	}
	return 0;
}

/*
* Gera uma palavra randomica de tamanho 'tam'
*/
__device__
void geraPalavraCuda(Palavra palGerada, unsigned int tam, hiprandState* estado)
{
	unsigned int i;
	// ID global da thread
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// Estado local
	hiprandState estadoLocal = estado[idx];

	for(i = 0; i < tam + 1; i++)
		palGerada[i] = hiprand(&estadoLocal) % 26 + 97;

	// Insere '\0'
	palGerada[tam] = '\0';
	estado[idx] = estadoLocal;
}

/*
* Verifica se palavra está no dicionário por busca binária
* se encontra verifica (atomicamente) se já foi encontrada anteriormente
*/
__device__
void verificaPalavraCuda(Palavra palGerada, palavraStruct *dic, unsigned int dicTam, unsigned int* cont)
{
	// Inicializa a busca binária
	unsigned int ini = 0, fim = dicTam - 1, med;
	int aux;
	// Enquanto for um intervalo válido
	while(ini <= fim)
	{
		// Pega o meio
		med = (ini + fim)/2;

		// Compara com a palavra do meio
		aux = comparaCuda(palGerada, dic[med].pal);

		// Se for menor
		if(aux < 0)
	    		fim = med - 1;
		// Se for maior
		else if(aux > 0)
    			ini = med + 1;
		// Se acertou
		else
		{
			if(!atomicCAS(&(dic[med].encontrado), 0, 1))
				atomicInc(cont, dicTam + 1);
			break;
		}
	}
}

/*
* Processo que cada thread CUDA faz, gera uma palavra de 1,2,3,4 e 5 letras
* e verifica se já foi gerada no dicionário
*/
__global__
void processoCuda(palavraStruct *dic, unsigned int dicTam, hiprandState *estado, unsigned int *cont)
{
	unsigned int i;
	for(i = 1 ; i < PTAM + 1 ; i++)
	{
		// Gera uma palavra aleatória
		Palavra palGerada;
		geraPalavraCuda(palGerada, i, estado);
		// Verifica quantas acertaram
		verificaPalavraCuda(palGerada, dic, dicTam, cont);
	}
}

/*
* Conta o número de linhas (palavras) do arquivo filtrado
*/
int nrLinhas(char *arquivo)
{
	char c;
	int cont = 0;
	FILE* ent;

	ent = fopen(arquivo,"r");
	do
	{
		c = fgetc(ent);
		if (c == (int) '\n' || c == (int) '\r')
			cont++;
	}while (c != EOF);
	fclose(ent);
	return cont;
}

/*
* Compara as palavras para ordenar o dicionário
*/
int comparaQuick(const void *a, const void *b) 
{ 
	struct palavraStruct *ia = (struct palavraStruct *)a;
	struct palavraStruct *ib = (struct palavraStruct *)b;
	return strcmp(ia->pal, ib->pal);
}

